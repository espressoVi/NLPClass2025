
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10000


void fill_matrix_random(float* matrix, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            int index = i * width + j;
            matrix[index] = (float)rand() / RAND_MAX * 1.0f;
        }
    }
}

__global__ void matrix_multiply_kernel(float* A, float* B, float* result, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width){
        float res = 0.0f;
        for (int k = 0; k < width; k++) {
            res += A[row * width + k] * B[k * width + col];
        }
        result[row * width + col] = res;
    }
}


int main() {
    clock_t start_time, end_time;
    srand(time(NULL));
    
    // Allocate matrices as 1D arrays
    float* matrixA = (float*)malloc(N * N * sizeof(float));
    float* matrixB = (float*)malloc(N * N * sizeof(float));
    float* result = (float*)malloc(N * N * sizeof(float));
    
    if (matrixA == NULL || matrixB == NULL || result == NULL) {
        printf("Failed!\n"); return 1;
    }
    
    // Fill matrices with random values
    fill_matrix_random(matrixA, N);
    fill_matrix_random(matrixB, N);

    // Allocate memory on GPU.
    float *Ad, *Bd, *Rd;
    int size = N * N * sizeof(float);
    hipMalloc((void**)&Ad, size);
    hipMalloc((void**)&Bd, size);
    hipMalloc((void**)&Rd, size);

    // Move objects to GPU.
    hipMemcpy(Ad, matrixA, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, matrixB, size, hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    // Size of each block (maximum 1024 threads per block). Set as multiple of 32 (warp scheduler)
    dim3 blockSize(32, 32);
    // Grid of blocks 63 x 63 as 63 * 16 = 1008 > 1000
    dim3 gridSize(
        (N + blockSize.x - 1) / blockSize.x,
        (N + blockSize.y - 1) / blockSize.y
    );


    // Multiply matrices
    start_time = clock();
    // __managed__

    matrix_multiply_kernel<<<gridSize, blockSize>>>(Ad, Bd, Rd, N);
    //cudaDeviceSynchronize();
    end_time = clock();

    // Copy result back to Host.
    hipMemcpy(result, Rd, size, hipMemcpyDeviceToHost);

    // Release memory
    hipFree(Ad); hipFree(Bd); hipFree(Rd);
    
    // Measure time
    double time_taken = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("Time: %f seconds\n", time_taken);
    
    // Free allocated memory
    free(matrixA);
    free(matrixB);
    free(result);
    
    return 0;
}
